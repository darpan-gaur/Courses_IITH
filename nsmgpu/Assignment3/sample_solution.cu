#include "hip/hip_runtime.h"
//to run it
// /usr/local/cuda-10.2/bin/nvcc PageRank.cu -arch=sm_70 -rdc=true
//using global synchrnization

#include<stdio.h>
#include<stdlib.h>
#include<limits.h>
#include<cmath>
#include<algorithm>
#include<hip/hip_runtime.h>
#include<hip/hip_cooperative_groups.h>
#include"helper.hpp"

namespace cg = cooperative_groups;



__device__ float diff ;
 __global__  void Compute_PR_Kernel(int * gpu_rev_OA, int * gpu_OA, int * gpu_srcList , float * gpu_node_pr , int V, int E , float beta, float delta, int maxIter) 
{
      diff =0.0f;
      unsigned int id = threadIdx.x + (blockDim.x * blockIdx.x);
      float sum=0.0f;
      int iterCount=0;
      
      auto g = cg::this_grid();
      
      
      if (id < V) 
      {
      

      for(int edge= gpu_rev_OA[id] ;edge< gpu_rev_OA[id+1];edge++)
         {
          
           int nbr=  gpu_srcList[edge];
           sum =sum + gpu_node_pr[nbr]/(gpu_OA[nbr+1]- gpu_OA[nbr]);
           
         }
         
         g.sync();
         
         
       
         float val=(1-delta)/V + delta * sum;
         float temp = std::fabs(val-gpu_node_pr[id]);
         atomicAdd(&diff,temp);
         gpu_node_pr[id]=val;
        
}

}


void Compute_PR(int * rev_OA, int * OA, int * cpu_srcList , float * node_pr , int V, int E)
{
  
  int    *gpu_rev_OA;
  int    *gpu_srcList;
  int    * gpu_OA;
  float  *gpu_node_pr;
  
  
  hipMalloc( &gpu_rev_OA, sizeof(int) * (1+V) ); //rev_OA
  hipMalloc( &gpu_OA, sizeof(int) * (1+V) );   //OA
  hipMalloc( &gpu_srcList, sizeof(int) * (E) ); //nbr
  hipMalloc( &gpu_node_pr, sizeof(float) * (V) ); //output
  
  
  unsigned int block_size;
	unsigned int num_blocks;
 
   for(int i=0; i< V; i++)
     {
         node_pr[i]= 1.0/V;
     }
   
  
  if(V <= 1024)
	{
		block_size = V;
		num_blocks = 1;
	}
	else
	{
		block_size = 1024;
		num_blocks = ceil(((float)V) / block_size);
			
	}
 
  hipMemcpy(gpu_rev_OA, rev_OA, sizeof(int) * (1+V), hipMemcpyHostToDevice);
  hipMemcpy(gpu_OA, OA, sizeof(int) * (1+V), hipMemcpyHostToDevice);
  hipMemcpy(gpu_srcList, cpu_srcList, sizeof(int) * (E), hipMemcpyHostToDevice);
  hipMemcpy(gpu_node_pr, node_pr , sizeof(float) * (V), hipMemcpyHostToDevice);
  
  
  
  float beta = 0.001;
  float delta = 0.85;
  int maxIter = 100;
  
  int iterCount=0;
  float diff_check;
 
  do
  {
  
  Compute_PR_Kernel<<<num_blocks , block_size>>>(gpu_rev_OA, gpu_OA, gpu_srcList, gpu_node_pr , V,E,  0.001, 0.85, 100);
  hipDeviceSynchronize();
  
  hipMemcpyFromSymbol(&diff_check, HIP_SYMBOL(diff), sizeof(float));
  iterCount=iterCount+1;
  
  }while ((diff_check>beta) &&(iterCount < maxIter));

  
  hipMemcpy(node_pr,gpu_node_pr , sizeof(float) * (V), hipMemcpyDeviceToHost);
  
  //output
  char *outputfilename = "outputN.txt";
  FILE *outputfilepointer;
  outputfilepointer = fopen(outputfilename, "w");

  for (int i = 0; i < V; i++)
  {
    fprintf(outputfilepointer, "%d  %0.9lf\n", i, node_pr[i]);
  }
 }

 int main(int argc , char ** argv)
{

  graph G(argv[1]);
  G.parseGraph();
  
  int V = G.num_nodes();
  int E = G.num_edges();
  

  float* node_pr;
  int *rev_OA;
  int *OA;
  int *cpu_srcList;
  
  
  
  node_pr = (float *)malloc( (V)*sizeof(float));
  rev_OA = (int *)malloc( (V+1)*sizeof(int));
  OA = (int *)malloc( (V+1)*sizeof(int));
  cpu_srcList = (int *)malloc( (E)*sizeof(int));
  
    
  for(int i=0; i<= V; i++) {
    int temp = G.rev_indexofNodes[i];
    rev_OA[i] = temp;
  }
  
   
  
   for(int i=0; i< E; i++) {
    int temp = G.srcList[i];
    cpu_srcList[i] = temp;
  }
  
 
  
  
   for(int i=0; i<= V; i++) {
    int temp = G.indexofNodes[i];
    OA[i] = temp;
  }
  
  Compute_PR(rev_OA, OA, cpu_srcList , node_pr , V, E);

 

}