#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
using namespace std;

ofstream outfile; //the handle for printing the output

// complete the following kernel... (1 d grid and 1 d block)
__global__ void per_row_column_kernel(long int *A, long int *B, long int *C,long int m, long int n, long int * sum , long int * sub){


  int row = (blockIdx.x * blockDim.x) + (threadIdx.x); // This is a unique thread ID formula for 1D grid with 1D blocks.
  
	if(row < m)
	{
			for(int col = 0; col < n; col++)
			{

					sum[row * n + col] = (A[row*n + col] + B[row*n + col]);
                    sub[row * n + col] = (B[row*n + col] - A[row*n + col]);
					C[col *m + row] = sum[row * n + col] - sub[row * n + col];
			}
	}

}

// complete the following kernel...(1d grid and 2 d blocks)
__global__ void per_column_row_kernel(long int *A, long int *B, long int *C,long int m, long int n, long int * sum , long int * sub){

  int col =  (blockIdx.x * blockDim.x * blockDim.y) + (threadIdx.x * blockDim.y) + threadIdx.y; // This is a unique thread ID formula for 1D grid with 2D blocks.
	if(col < n)
	{
			for(int row = 0; row < m; row++)
			{
                sum[row * n + col] = (A[row*n + col] + B[row*n + col]);
                sub[row * n + col] = (B[row*n + col] - A[row*n + col]);
				C[col *m + row] = sum[row * n + col] - sub[row * n + col];
			}
	}

}

// complete the following kernel...(2d grid and 2 d blocks)
__global__ void per_element_kernel(long int *A, long int *B, long int *C,long int m, long int n, long int * sum , long int * sub){

  int tid = (blockIdx.x * gridDim.y * blockDim.x* blockDim.y) + (blockIdx.y * blockDim.x * blockDim.y) +(threadIdx.x * blockDim.y) + threadIdx.y; 

	// Extract the row-col IDs corresponding to the unique thread ID
	int row = tid / n;
	int col = tid % n;

	if(row < m && col < n)
	{
		    sum[row * n + col] = (A[row*n + col] + B[row*n + col]);
			sub[row * n + col] = (B[row*n + col] - A[row*n + col]);
			C[col *m + row] = sum[row * n + col] - sub[row * n + col];
	}
  

}

/**
 * Prints any 1D array in the form of a matrix 
 * */
void printMatrix(long int *arr, long int rows, long int cols, char* filename) {

	outfile.open(filename);
	for(long int i = 0; i < cols; i++) {
		for(long int j = 0; j < rows; j++) {
			outfile<<arr[i * rows + j]<<" ";
		}
		outfile<<"\n";
	}
	outfile.close();
}

int main(int argc,char **argv){

	//variable declarations
	long int m,n;	
	cin>>m>>n;	


	//host_arrays 
	long int *h_a,*h_b,*h_c;

	//device arrays 
	long int *d_a,*d_b,*d_c;
 
 //temporary device array for addition and subtraction
 long int *d_sum, *d_sub;
	
	//Allocating space for the host_arrays 
	h_a = (long int *) malloc(m * n * sizeof(long int));
	h_b = (long int *) malloc(m * n * sizeof(long int));	
	h_c = (long int *) malloc(m * n * sizeof(long int));	

	//Allocating memory for the device arrays 
	hipMalloc(&d_a, m * n * sizeof(long int));
	hipMalloc(&d_b, m * n * sizeof(long int));
	hipMalloc(&d_c, m * n * sizeof(long int));
  hipMalloc(&d_sum, m * n * sizeof(long int));
  hipMalloc(&d_sub, m * n * sizeof(long int));

	//Read the input matrix A 
	for(long int i = 0; i < m * n; i++) {
		cin>>h_a[i];
	}

	//Read the input matrix B 
	for(long int i = 0; i < m * n; i++) {
		cin>>h_b[i];
	}

	//Transfer the input host arrays to the device 
	hipMemcpy(d_a, h_a, m * n * sizeof(long int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, m * n * sizeof(long int), hipMemcpyHostToDevice);

	long int gridDimx, gridDimy;
	//Launch the kernels 
	/**
	 * Kernel 1 - per_row_column_kernel
	 * To be launched with 1D grid, 1D block
	 * */
	gridDimx = ceil(float(m) / 1024);
	dim3 grid1(gridDimx,1,1);
	dim3 block1(1024,1,1);
	per_row_column_kernel<<<grid1,block1>>>(d_a,d_b,d_c,m,n,d_sum,d_sub);
	hipDeviceSynchronize();
	hipMemcpy(h_c, d_c, m * n * sizeof(long int), hipMemcpyDeviceToHost);
	printMatrix(h_c, m, n,"kernel1.txt");
	
	/**
	 * Kernel 2 - per_column_row_kernel
	 * To be launched with 1D grid, 2D block
	 * */
	gridDimx = ceil(float(n) / 1024);
	dim3 grid2(gridDimx,1,1);
	dim3 block2(32,32,1);
	per_column_row_kernel<<<grid2,block2>>>(d_a,d_b,d_c,m,n,d_sum,d_sub);
	hipDeviceSynchronize();
	hipMemcpy(h_c, d_c, m * n * sizeof(long int), hipMemcpyDeviceToHost);
	printMatrix(h_c, m, n,"kernel2.txt");

	/**
	 * Kernel 3 - per_element_kernel
	 * To be launched with 2D grid, 2D block
	 * */
	gridDimx = ceil(float(n) / 16);
	gridDimy = ceil(float(m) / 64);
	dim3 grid3(gridDimx,gridDimy,1);
	dim3 block3(64,16,1);
	per_element_kernel<<<grid3,block3>>>(d_a,d_b,d_c,m,n,d_sum,d_sub);
	hipDeviceSynchronize();
	hipMemcpy(h_c, d_c, m * n * sizeof(long int), hipMemcpyDeviceToHost);
	printMatrix(h_c, m, n,"kernel3.txt");


	return 0;
}
